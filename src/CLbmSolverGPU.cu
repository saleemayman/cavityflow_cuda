#include "hip/hip_runtime.h"
/*
 * Copyright
 * 2010 Martin Schreiber
 * 2013 Arash Bakhtiari
 * 2016 Christoph Riesinger, Ayman Saleem
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "CLbmSolverGPU.cuh"

#include <cassert>

#include "gpukernels/lbm_alpha.cuh"
#include "gpukernels/lbm_beta.cuh"
#include "gpukernels/lbm_init.cuh"

template <class T>
CLbmSolverGPU<T>::CLbmSolverGPU(
        int id,
        std::vector<dim3> threadsPerBlock,
        CVector<3, T> &globalLength,
        CDomain<T> &domain,
        std::vector<Flag> boundaryConditions,
        T timestepSize,
        CVector<3, T> &velocity,
        CVector<3, T> &acceleration,
        T viscosity,
        T maxVelocityDimLess,
        T maxAccelerationDimLess,
        bool storeDensities,
        bool storeVelocities,
        bool doLogging) :
        CLbmSolver<T>(id, globalLength,
                domain, boundaryConditions,
                timestepSize, velocity, acceleration,
                viscosity, maxVelocityDimLess, maxAccelerationDimLess,
                storeDensities, storeVelocities, doLogging),
        threadsPerBlock(threadsPerBlock)
{
    int numOfGPUsPerNode;

    GPU_ERROR_CHECK(hipGetDeviceCount(&numOfGPUsPerNode))
    GPU_ERROR_CHECK(hipSetDevice(this->id % numOfGPUsPerNode))

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::CLbmSolverGPU() -----" << std::endl;
        std::cout << "id:                                                 " << this->id << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
        std::cout << "number of GPUs per node:                            " << numOfGPUsPerNode << std::endl;
        std::cout << "number of selected GPU:                             " << (this->id % numOfGPUsPerNode) << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
    }

    GPU_ERROR_CHECK(hipMalloc(&densityDistributions, NUM_LATTICE_VECTORS * this->domain.getNumOfCellsWithHalo() * sizeof(T)))
    GPU_ERROR_CHECK(hipMalloc(&flags, this->domain.getNumOfCellsWithHalo() * sizeof(Flag)))
    if(this->storeDensities)
        GPU_ERROR_CHECK(hipMalloc(&densities, this->domain.getNumOfCellsWithHalo() * sizeof(T)))
    if(this->storeVelocities)
        GPU_ERROR_CHECK(hipMalloc(&velocities, 3 * this->domain.getNumOfCellsWithHalo() * sizeof(T)))

    if (doLogging) {
        std::cout << "size of allocated memory for density distributions: " << ((T)(NUM_LATTICE_VECTORS * this->domain.getNumOfCellsWithHalo() * sizeof(T)) / (T)(1<<20)) << " MBytes" << std::endl;
        std::cout << "size of allocated memory for flags:                 " << ((T)(this->domain.getNumOfCellsWithHalo() * sizeof(Flag)) / (T)(1<<20)) << " MBytes" << std::endl;
        if(this->storeDensities)
            std::cout << "size of allocated memory for velocities:            " << ((T)(3 * this->domain.getNumOfCellsWithHalo() * sizeof(T)) / (T)(1<<20)) << " MBytes" << std::endl;
        if(this->storeVelocities)
            std::cout << "size of allocated memory for densities:             " << ((T)(this->domain.getNumOfCellsWithHalo() * sizeof(T)) / (T)(1<<20)) << " MBytes" << std::endl;
    }

    dim3 blocksPerGrid = getBlocksPerGrid(3, this->domain.getSizeWithHalo(), this->threadsPerBlock[0]);

    if (doLogging) {
        std::cout << "threads per block:                                  [" << this->threadsPerBlock[0].x << ", " << this->threadsPerBlock[0].y << ", " << this->threadsPerBlock[0].z << "]" << std::endl;
        std::cout << "blocks per grid:                                    [" << blocksPerGrid.x << ", " << blocksPerGrid.y << ", " << blocksPerGrid.z << "]" << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }

    lbm_init<T><<<blocksPerGrid, this->threadsPerBlock[0]>>>(
        densityDistributions,
        flags,
        velocities,
        densities,
        boundaryConditions[0],
        boundaryConditions[1],
        boundaryConditions[2],
        boundaryConditions[3],
        boundaryConditions[4],
        boundaryConditions[5],
        velocityDimLess[0],
        domain.getSizeWithHalo()[0],
        domain.getSizeWithHalo()[1],
        domain.getSizeWithHalo()[2],
        storeDensities,
        storeVelocities);
    GPU_ERROR_CHECK(hipPeekAtLastError())
    
    if (doLogging) {
        std::cout << "Domain successfully initialized." << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
    }
}

template <class T>
CLbmSolverGPU<T>::~CLbmSolverGPU()
{
    if(storeVelocities)
        GPU_ERROR_CHECK(hipFree(velocities))
    if(storeDensities)
        GPU_ERROR_CHECK(hipFree(densities))
    GPU_ERROR_CHECK(hipFree(flags))
    GPU_ERROR_CHECK(hipFree(densityDistributions))
}

template <class T>
void CLbmSolverGPU<T>::simulationStepAlpha(hipStream_t* stream)
{
    dim3 blocksPerGrid = getBlocksPerGrid(3, domain.getSizeWithHalo(), threadsPerBlock[1]);

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::simulationStepAlpha() -----" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "---------------------------------------------------" << std::endl;
        std::cout << "threads per block: [" << threadsPerBlock[1].x << ", " << threadsPerBlock[1].y << ", " << threadsPerBlock[1].z << "]" << std::endl;
        std::cout << "blocks per grid:   [" << blocksPerGrid.x << ", " << blocksPerGrid.y << ", " << blocksPerGrid.z << "]" << std::endl;
        std::cout << "---------------------------------------------------" << std::endl;
    }

    lbm_kernel_alpha<T><<<blocksPerGrid, threadsPerBlock[1], 0, ((stream == NULL) ? 0 : *stream)>>>(
            densityDistributions,
            flags,
            velocities,
            densities,
            tauInv,
            accelerationDimLess[0],
            accelerationDimLess[1],
            accelerationDimLess[2],
            velocityDimLess[0],
            0,
            0,
            0,
            domain.getSizeWithHalo()[0],
            domain.getSizeWithHalo()[1],
            domain.getSizeWithHalo()[2],
            domain.getSizeWithHalo()[0],
            domain.getSizeWithHalo()[1],
            domain.getSizeWithHalo()[2],
            storeDensities,
            storeVelocities);
    GPU_ERROR_CHECK(hipPeekAtLastError())

    if (doLogging)
    {
        std::cout << "Alpha kernel was successfully executed on the whole subdomain." << std::endl;
        std::cout << "---------------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::simulationStepAlpha()
{
    simulationStepAlpha(NULL);
}

template <class T>
void CLbmSolverGPU<T>::simulationStepAlpha(CVector<3, int> origin, CVector<3, int> size, hipStream_t* stream)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    dim3 blocksPerGrid = getBlocksPerGrid(3, size, threadsPerBlock[1]);

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::simulationStepAlpha() -----" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "---------------------------------------------------" << std::endl;
        std::cout << "threads per block: [" << threadsPerBlock[1].x << ", " << threadsPerBlock[1].y << ", " << threadsPerBlock[1].z << "]" << std::endl;
        std::cout << "blocks per grid:   [" << blocksPerGrid.x << ", " << blocksPerGrid.y << ", " << blocksPerGrid.z << "]" << std::endl;
        std::cout << "---------------------------------------------------" << std::endl;
    }

    lbm_kernel_alpha<T><<<blocksPerGrid, threadsPerBlock[1], 0, ((stream == NULL) ? 0 : *stream)>>>(
            densityDistributions,
            flags,
            velocities,
            densities,
            tauInv,
            accelerationDimLess[0],
            accelerationDimLess[1],
            accelerationDimLess[2],
            velocityDimLess[0],
            origin[0],
            origin[1],
            origin[2],
            size[0],
            size[1],
            size[2],
            domain.getSizeWithHalo()[0],
            domain.getSizeWithHalo()[1],
            domain.getSizeWithHalo()[2],
            storeDensities,
            storeVelocities);
    GPU_ERROR_CHECK(hipPeekAtLastError())

    if (doLogging)
    {
        std::cout << "Alpha kernel was successfully executed on the following subdomain:" << std::endl;
        std::cout << "origin:            " << origin << std::endl;
        std::cout << "size:              " << size << std::endl;
        std::cout << "---------------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::simulationStepAlpha(CVector<3, int> origin, CVector<3, int> size)
{
    simulationStepAlpha(origin, size, NULL);
}

template <class T>
void CLbmSolverGPU<T>::simulationStepBeta(hipStream_t* stream)
{
    dim3 blocksPerGrid = getBlocksPerGrid(3, domain.getSizeWithHalo(), threadsPerBlock[2]);
    size_t sMemSize = 12 * sizeof(T) * getSize(threadsPerBlock[2]);

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::simulationStepBeta() -----" << std::endl;
        std::cout << "id:                 " << id << std::endl;
        std::cout << "--------------------------------------------------" << std::endl;
        std::cout << "threads per block:  [" << threadsPerBlock[2].x << ", " << threadsPerBlock[2].y << ", " << threadsPerBlock[2].z << "]" << std::endl;
        std::cout << "blocks per grid:    [" << blocksPerGrid.x << ", " << blocksPerGrid.y << ", " << blocksPerGrid.z << "]" << std::endl;
        std::cout << "shared memory size: " << ((T)sMemSize / (T)(1<<10)) << " KB" << std::endl;
        std::cout << "--------------------------------------------------" << std::endl;
    }

    lbm_kernel_beta<T><<<blocksPerGrid, threadsPerBlock[2], sMemSize, ((stream == NULL) ? 0 : *stream)>>>(
            densityDistributions,
            flags,
            velocities,
            densities,
            tauInv,
            accelerationDimLess[0],
            accelerationDimLess[1],
            accelerationDimLess[2],
            velocityDimLess[0],
            0,
            0,
            0,
            domain.getSizeWithHalo()[0],
            domain.getSizeWithHalo()[1],
            domain.getSizeWithHalo()[2],
            domain.getSizeWithHalo()[0],
            domain.getSizeWithHalo()[1],
            domain.getSizeWithHalo()[2],
            getSize(threadsPerBlock[2]),
            isPowerOfTwo(domain.getNumOfCellsWithHalo()),
            isPowerOfTwo(getSize(threadsPerBlock[2])),
            storeDensities,
            storeVelocities);
    GPU_ERROR_CHECK(hipPeekAtLastError())

    if (doLogging)
    {
        std::cout << "Beta kernel was successfully executed on the whole subdomain." << std::endl;
        std::cout << "--------------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::simulationStepBeta()
{
    simulationStepBeta(NULL);
}

template <class T>
void CLbmSolverGPU<T>::simulationStepBeta(CVector<3, int> origin, CVector<3, int> size, hipStream_t* stream)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    dim3 blocksPerGrid = getBlocksPerGrid(3, size, threadsPerBlock[2]);
    size_t sMemSize = 12 * sizeof(T) * getSize(threadsPerBlock[2]);

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::simulationStepBeta() -----" << std::endl;
        std::cout << "id:                 " << id << std::endl;
        std::cout << "--------------------------------------------------" << std::endl;
        std::cout << "threads per block:  [" << threadsPerBlock[2].x << ", " << threadsPerBlock[2].y << ", " << threadsPerBlock[2].z << "]" << std::endl;
        std::cout << "blocks per grid:    [" << blocksPerGrid.x << ", " << blocksPerGrid.y << ", " << blocksPerGrid.z << "]" << std::endl;
        std::cout << "shared memory size: " << ((T)sMemSize / (T)(1<<10)) << " KB" << std::endl;
        std::cout << "--------------------------------------------------" << std::endl;
    }

    lbm_kernel_beta<T><<<blocksPerGrid, threadsPerBlock[2], sMemSize, ((stream == NULL) ? 0 : *stream)>>>(
            densityDistributions,
            flags,
            velocities,
            densities,
            tauInv,
            accelerationDimLess[0],
            accelerationDimLess[1],
            accelerationDimLess[2],
            velocityDimLess[0],
            origin[0],
            origin[1],
            origin[2],
            size[0],
            size[1],
            size[2],
            domain.getSizeWithHalo()[0],
            domain.getSizeWithHalo()[1],
            domain.getSizeWithHalo()[2],
            getSize(threadsPerBlock[2]),
            isPowerOfTwo(domain.getNumOfCellsWithHalo()),
            isPowerOfTwo(getSize(threadsPerBlock[2])),
            storeDensities,
            storeVelocities);
    GPU_ERROR_CHECK(hipPeekAtLastError())

    if (doLogging)
    {
        std::cout << "Beta kernel was successfully executed on the following subdomain." << std::endl;
        std::cout << "origin:             " << origin << std::endl;
        std::cout << "size:               " << size << std::endl;
        std::cout << "--------------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::simulationStepBeta(CVector<3, int> origin, CVector<3, int> size)
{
    simulationStepBeta(origin, size, NULL);
}

template <class T>
void CLbmSolverGPU<T>::getDensityDistributions(CVector<3, int> &origin, CVector<3, int> &size, T* hDensityDistributions, hipStream_t* stream)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    hipMemcpy3DParms params = {0};

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::getDensityDistributions() -----" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "domain origin:     " << domain.getOrigin() << std::endl;
        std::cout << "domain size:       " << domain.getSize() << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "cuboid origin:     " << origin << std::endl;
        std::cout << "cuboid size:       " << size << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }

    for(int latticeVector = 0; latticeVector < NUM_LATTICE_VECTORS; latticeVector++)
    {
        // domain location and size
        params.srcPtr = make_hipPitchedPtr(&densityDistributions[latticeVector * domain.getNumOfCellsWithHalo()], domain.getSizeWithHalo()[0] * sizeof(T), domain.getSizeWithHalo()[0], domain.getSizeWithHalo()[1]);
        // cuboid origin
        params.srcPos = make_hipPos(origin[0] * (sizeof(T) / sizeof(unsigned char)), origin[1], origin[2]);
        // hDensityDistributions location and size
        params.dstPtr = make_hipPitchedPtr(&hDensityDistributions[latticeVector * size.elements()], size[0] * sizeof(T), size[0], size[1]);
        // hDensityDistributions origin
        params.dstPos = make_hipPos(0, 0, 0);
        // cuboid size
        params.extent = make_hipExtent(size[0] * (sizeof(T) / sizeof(unsigned char)), size[1], size[2]);
        params.kind = hipMemcpyDeviceToHost;

        GPU_ERROR_CHECK(hipMemcpy3DAsync(&params, (stream == NULL) ? 0 : *stream))
    }

    if (doLogging)
    {
        std::cout << "A copy operation from device to host was performed." << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::getDensityDistributions(CVector<3, int>& origin, CVector<3, int>& size, T* hDensityDistributions)
{
    getDensityDistributions(origin, size, hDensityDistributions, NULL);
}

template <class T>
void CLbmSolverGPU<T>::getDensityDistributions(T* hDensityDistributions, hipStream_t* stream)
{
    CVector<3, int> origin(1);
    CVector<3, int> size(domain.getSize());

    getDensityDistributions(origin, size, hDensityDistributions, stream);
}

template <class T>
void CLbmSolverGPU<T>::getDensityDistributions(T* hDensityDistributions)
{
    getDensityDistributions(hDensityDistributions, NULL);
}

template <class T>
void CLbmSolverGPU<T>::setDensityDistributions(CVector<3, int> &origin, CVector<3, int> &size, Direction direction, T* hDensityDistributions, hipStream_t* stream)
{
    assert(0 <= direction < 6);
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    CVector<3, int> norm(0);
    hipMemcpy3DParms params = {0};

    switch(direction)
    {
    case LEFT:
        norm[0] = 1;
        break;
    case RIGHT:
        norm[0] = -1;
        break;
    case BOTTOM:
        norm[1] = 1;
        break;
    case TOP:
        norm[1] = -1;
        break;
    case BACK:
        norm[2] = 1;
        break;
    case FRONT:
        norm[2] = -1;
        break;
    }

    if (doLogging) {
        std::cout << "----- CLbmSolverGPU<T>::setDensityDistributions() -----" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "domain origin:     " << domain.getOrigin() << std::endl;
        std::cout << "domain size:       " << domain.getSize() << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "cuboid origin:     " << origin << std::endl;
        std::cout << "cuboid size:       " << size << std::endl;
        std::cout << "direction:         " << norm << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }

    for (int latticeVector = 0; latticeVector < NUM_LATTICE_VECTORS; latticeVector++)
    {
        if(norm.dotProd(lbm_units[latticeVector]) > 0)
        {
            // hDensityDistributions location and size
            params.srcPtr = make_hipPitchedPtr(&hDensityDistributions[latticeVector * size.elements()], size[0] * sizeof(T), size[0], size[1]);
            // hDensityDistributions origin
            params.srcPos = make_hipPos(0, 0, 0);
            // domain location and size
            params.dstPtr = make_hipPitchedPtr(&densityDistributions[latticeVector * domain.getNumOfCellsWithHalo()], domain.getSizeWithHalo()[0] * sizeof(T), domain.getSizeWithHalo()[0], domain.getSizeWithHalo()[1]);
            // cuboid origin
            params.dstPos = make_hipPos(origin[0] * (sizeof(T) / sizeof(unsigned char)), origin[1], origin[2]);
            // cuboid size
            params.extent = make_hipExtent(size[0] * (sizeof(T) / sizeof(unsigned char)), size[1], size[2]);
            params.kind = hipMemcpyHostToDevice;

            GPU_ERROR_CHECK(hipMemcpy3DAsync(&params, (stream == NULL) ? 0 : *stream))
        }
    }

    if (doLogging) {
        std::cout << "A copy operation from host to device for lattice vectors in direction " << direction << " was performed." << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::setDensityDistributions(CVector<3, int>& origin, CVector<3, int>& size, Direction direction, T* hDensityDistributions)
{
    setDensityDistributions(origin, size, direction, hDensityDistributions, NULL);
}

template <class T>
void CLbmSolverGPU<T>::setDensityDistributions(CVector<3, int> &origin, CVector<3, int> &size, T* hDensityDistributions, hipStream_t* stream)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    hipMemcpy3DParms params = {0};

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::setDensityDistributions() -----" << std::endl;
        std::cout << "A copy operation from host to device was performed." << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "domain origin:     " << domain.getOrigin() << std::endl;
        std::cout << "domain size:       " << domain.getSize() << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "cuboid origin:     " << origin << std::endl;
        std::cout << "cuboid size:       " << size << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }

    for (int latticeVector = 0; latticeVector < NUM_LATTICE_VECTORS; latticeVector++)
    {
        // hDensityDistributions location and size
        params.srcPtr = make_hipPitchedPtr(&hDensityDistributions[latticeVector * size.elements()], size[0] * sizeof(T), size[0], size[1]);
        // hDensityDistributions origin
        params.srcPos = make_hipPos(0, 0, 0);
        // domain location and size
        params.dstPtr = make_hipPitchedPtr(&densityDistributions[latticeVector * domain.getNumOfCellsWithHalo()], domain.getSizeWithHalo()[0] * sizeof(T), domain.getSizeWithHalo()[0], domain.getSizeWithHalo()[1]);
        // cuboid origin
        params.dstPos = make_hipPos(origin[0] * (sizeof(T) / sizeof(unsigned char)), origin[1], origin[2]);
        // cuboid size
        params.extent = make_hipExtent(size[0] * (sizeof(T) / sizeof(unsigned char)), size[1], size[2]);
        params.kind = hipMemcpyHostToDevice;

        GPU_ERROR_CHECK(hipMemcpy3DAsync(&params, (stream == NULL) ? 0 : *stream))
    }

    if (doLogging)
    {
        std::cout << "A copy operation from host to device was performed." << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::setDensityDistributions(CVector<3, int>& origin, CVector<3, int>& size, T* hDensityDistributions)
{
    setDensityDistributions(origin, size, hDensityDistributions, NULL);
}

template <class T>
void CLbmSolverGPU<T>::setDensityDistributions(T* hDensityDistributions, hipStream_t* stream)
{
    CVector<3, int> origin(1);
    CVector<3, int> size(domain.getSize());

    setDensityDistributions(origin, size, hDensityDistributions, stream);
}

template <class T>
void CLbmSolverGPU<T>::setDensityDistributions(T* hDensityDistributions)
{
    setDensityDistributions(hDensityDistributions, NULL);
}

template <class T>
void CLbmSolverGPU<T>::getFlags(CVector<3, int> &origin, CVector<3, int> &size, Flag* hFlags)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    hipMemcpy3DParms params = {0};

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::getFlags() -----" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "domain origin:     " << domain.getOrigin() << std::endl;
        std::cout << "domain size:       " << domain.getSize() << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "cuboid origin:     " << origin << std::endl;
        std::cout << "cuboid size:       " << size << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }

    // domain location and size
    params.srcPtr = make_hipPitchedPtr(flags, domain.getSizeWithHalo()[0] * sizeof(Flag), domain.getSizeWithHalo()[0], domain.getSizeWithHalo()[1]);
    // cuboid origin
    params.srcPos = make_hipPos(origin[0] * (sizeof(Flag) / sizeof(unsigned char)), origin[1], origin[2]);
    // hFlags location and size
    params.dstPtr = make_hipPitchedPtr(hFlags, size[0] * sizeof(Flag), size[0], size[1]);
    // hFlags origin
    params.dstPos = make_hipPos(0, 0, 0);
    // cuboid size
    params.extent = make_hipExtent(size[0] * (sizeof(Flag) / sizeof(unsigned char)), size[1], size[2]);
    params.kind = hipMemcpyDeviceToHost;

    GPU_ERROR_CHECK(hipMemcpy3D(&params))

    if (doLogging)
    {
        std::cout << "A copy operation from device to host was performed." << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::getFlags(Flag* hFlags)
{
    CVector<3, int> origin(1);
    CVector<3, int> size(domain.getSize());

    getFlags(origin, size, hFlags);
}

template <class T>
void CLbmSolverGPU<T>::setFlags(CVector<3, int> &origin, CVector<3, int> &size, Flag* hFlags)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    hipMemcpy3DParms params = {0};

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::setFlags() -----" << std::endl;
        std::cout << "A copy operation from host to device was performed." << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "domain origin:     " << domain.getOrigin() << std::endl;
        std::cout << "domain size:       " << domain.getSize() << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
        std::cout << "cuboid origin:     " << origin << std::endl;
        std::cout << "cuboid size:       " << size << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }

    // hFlags location and size
    params.srcPtr = make_hipPitchedPtr(hFlags, size[0] * sizeof(Flag), size[0], size[1]);
    // hFlags origin
    params.srcPos = make_hipPos(0, 0, 0);
    // domain location and size
    params.dstPtr = make_hipPitchedPtr(flags, domain.getSizeWithHalo()[0] * sizeof(Flag), domain.getSizeWithHalo()[0], domain.getSizeWithHalo()[1]);
    // cuboid origin
    params.dstPos = make_hipPos(origin[0] * (sizeof(Flag) / sizeof(unsigned char)), origin[1], origin[2]);
    // cuboid size
    params.extent = make_hipExtent(size[0] * (sizeof(Flag) / sizeof(unsigned char)), size[1], size[2]);
    params.kind = hipMemcpyHostToDevice;

    GPU_ERROR_CHECK(hipMemcpy3D(&params))

    if (doLogging)
    {
        std::cout << "A copy operation from host to device was performed." << std::endl;
        std::cout << "-------------------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::setFlags(Flag* hFlags)
{
    CVector<3, int> origin(1);
    CVector<3, int> size(domain.getSize());

    setFlags(origin, size, hFlags);
}

template <class T>
void CLbmSolverGPU<T>::getVelocities(CVector<3, int> &origin, CVector<3, int> &size, T* hVelocities)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    hipMemcpy3DParms params = {0};

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::getVelocities() -----" << std::endl;
        std::cout << "A copy operation from device to host was performed." << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
        std::cout << "domain origin:     " << domain.getOrigin() << std::endl;
        std::cout << "domain size:       " << domain.getSize() << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
        std::cout << "cuboid origin:     " << origin << std::endl;
        std::cout << "cuboid size:       " << size << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
    }

    for (int dim = 0; dim < 3; dim++)
    {
        // domain location and size
        params.srcPtr = make_hipPitchedPtr(&velocities[dim * domain.getNumOfCellsWithHalo()], domain.getSizeWithHalo()[0] * sizeof(T), domain.getSizeWithHalo()[0], domain.getSizeWithHalo()[1]);
        // cuboid origin
        params.srcPos = make_hipPos(origin[0] * (sizeof(T) / sizeof(unsigned char)), origin[1], origin[2]);
        // hVelocities location and size
        params.dstPtr = make_hipPitchedPtr(&hVelocities[dim * size.elements()], size[0] * sizeof(T), size[0], size[1]);
        // hVelocities origin
        params.dstPos = make_hipPos(0, 0, 0);
        // cuboid size
        params.extent = make_hipExtent(size[0] * (sizeof(T) / sizeof(unsigned char)), size[1], size[2]);
        params.kind = hipMemcpyDeviceToHost;

        GPU_ERROR_CHECK(hipMemcpy3D(&params))
    }

    if (doLogging)
    {
        std::cout << "A copy operation from device to host was performed." << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::getVelocities(T* hVelocities)
{
    CVector<3, int> origin(1);
    CVector<3, int> size(domain.getSize());

    getVelocities(origin, size, hVelocities);
}

template <class T>
void CLbmSolverGPU<T>::setVelocities(CVector<3, int> &origin, CVector<3, int> &size, T* hVelocities)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    hipMemcpy3DParms params = {0};

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::setVelocities() -----" << std::endl;
        std::cout << "A copy operation from host to device was performed." << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
        std::cout << "domain origin:     " << domain.getOrigin() << std::endl;
        std::cout << "domain size:       " << domain.getSize() << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
        std::cout << "cuboid origin:     " << origin << std::endl;
        std::cout << "cuboid size:       " << size << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
    }

    for (int dim = 0; dim < 3; dim++)
    {
        // hVelocities location and size
        params.srcPtr = make_hipPitchedPtr(&hVelocities[dim * size.elements()], size[0] * sizeof(T), size[0], size[1]);
        // hVelocities origin
        params.srcPos = make_hipPos(0, 0, 0);
        // domain location and size
        params.dstPtr = make_hipPitchedPtr(&velocities[dim * domain.getNumOfCellsWithHalo()], domain.getSizeWithHalo()[0] * sizeof(T), domain.getSizeWithHalo()[0], domain.getSizeWithHalo()[1]);
        // cuboid origin
        params.dstPos = make_hipPos(origin[0] * (sizeof(T) / sizeof(unsigned char)), origin[1], origin[2]);
        // cuboid size
        params.extent = make_hipExtent(size[0] * (sizeof(T) / sizeof(unsigned char)), size[1], size[2]);
        params.kind = hipMemcpyHostToDevice;

        GPU_ERROR_CHECK(hipMemcpy3D(&params))
    }

    if (doLogging)
    {
        std::cout << "A copy operation from host to device was performed." << std::endl;
        std::cout << "---------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::setVelocities(T* hVelocities)
{
    CVector<3, int> origin(1);
    CVector<3, int> size(domain.getSize());

    setVelocities(origin, size, hVelocities);
}

template <class T>
void CLbmSolverGPU<T>::getDensities(CVector<3, int> &origin, CVector<3, int> &size, T* hDensities)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    hipMemcpy3DParms params = {0};

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::getDensities() -----" << std::endl;
        std::cout << "A copy operation from device to host was performed." << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
        std::cout << "domain origin:     " << domain.getOrigin() << std::endl;
        std::cout << "domain size:       " << domain.getSize() << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
        std::cout << "cuboid origin:     " << origin << std::endl;
        std::cout << "cuboid size:       " << size << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
    }

    // domain location and size
    params.srcPtr = make_hipPitchedPtr(densities, domain.getSizeWithHalo()[0] * sizeof(T), domain.getSizeWithHalo()[0], domain.getSizeWithHalo()[1]);
    // cuboid origin
    params.srcPos = make_hipPos(origin[0] * (sizeof(T) / sizeof(unsigned char)), origin[1], origin[2]);
    // hDensities location and size
    params.dstPtr = make_hipPitchedPtr(hDensities, size[0] * sizeof(T), size[0], size[1]);
    // hDensities origin
    params.dstPos = make_hipPos(0, 0, 0);
    // cuboid size
    params.extent = make_hipExtent(size[0] * (sizeof(T) / sizeof(unsigned char)), size[1], size[2]);
    params.kind = hipMemcpyDeviceToHost;

    GPU_ERROR_CHECK(hipMemcpy3D(&params))

    if (doLogging)
    {
        std::cout << "A copy operation from device to host was performed." << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::getDensities(T* hDensities)
{
    CVector<3, int> origin(1);
    CVector<3, int> size(domain.getSize());

    getDensities(origin, size, hDensities);
}

template <class T>
void CLbmSolverGPU<T>::setDensities(CVector<3, int> &origin, CVector<3, int> &size, T* hDensities)
{
    assert(origin[0] >= 0 && origin[1] >= 0 && origin[2] >= 0);
    assert(size[0] > 0 && size[1] > 0 && size[2] > 0);
    assert(origin[0] + size[0] <= domain.getSizeWithHalo()[0]);
    assert(origin[1] + size[1] <= domain.getSizeWithHalo()[1]);
    assert(origin[2] + size[2] <= domain.getSizeWithHalo()[2]);

    hipMemcpy3DParms params = {0};

    if (doLogging)
    {
        std::cout << "----- CLbmSolverGPU<T>::setDensities() -----" << std::endl;
        std::cout << "A copy operation from host to device was performed." << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
        std::cout << "id:                " << id << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
        std::cout << "domain origin:     " << domain.getOrigin() << std::endl;
        std::cout << "domain size:       " << domain.getSize() << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
        std::cout << "cuboid origin:     " << origin << std::endl;
        std::cout << "cuboid size:       " << size << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
    }

    // hDensities location and size
    params.srcPtr = make_hipPitchedPtr(hDensities, size[0] * sizeof(T), size[0], size[1]);
    // hDensities origin
    params.srcPos = make_hipPos(0, 0, 0);
    // domain location and size
    params.dstPtr = make_hipPitchedPtr(densities, domain.getSizeWithHalo()[0] * sizeof(T), domain.getSizeWithHalo()[0], domain.getSizeWithHalo()[1]);
    // cuboid origin
    params.dstPos = make_hipPos(origin[0] * (sizeof(T) / sizeof(unsigned char)), origin[1], origin[2]);
    // cuboid size
    params.extent = make_hipExtent(size[0] * (sizeof(T) / sizeof(unsigned char)), size[1], size[2]);
    params.kind = hipMemcpyHostToDevice;

    GPU_ERROR_CHECK(hipMemcpy3D(&params))

    if (doLogging)
    {
        std::cout << "A copy operation from host to device was performed." << std::endl;
        std::cout << "--------------------------------------------" << std::endl;
    }
}

template <class T>
void CLbmSolverGPU<T>::setDensities(T* hDensities)
{
    CVector<3, int> origin(1);
    CVector<3, int> size(domain.getSize());

    setDensities(origin, size, hDensities);
}

template class CLbmSolverGPU<float>;
template class CLbmSolverGPU<double>;
