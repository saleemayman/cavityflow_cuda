#include "hip/hip_runtime.h"
/*
 * Copyright
 * 2010 Martin Schreiber
 * 2013 Arash Bakhtiari
 * 2016 Christoph Riesinger, Ayman Saleem
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hip/hip_vector_types.h"

#include "../common.h"

__device__ int3 getCubePosition(int linear_position, const int domainCells_x, const int domainCells_y)
{
    int3 pos;

    /*
     * TODO
     * use AND operation to speed up (but this function is only used during initialization)
     */
    pos.x = linear_position % domainCells_x;
    linear_position /= domainCells_x;

    pos.y = linear_position % domainCells_y;
    linear_position /= domainCells_y;

    pos.z = linear_position;// % CUBE_SIZE_Z;

    return pos;
}

template<typename T>
__global__ void init_kernel(
        T *global_dd,           // density distributions
        int *flags,             // flags
        T *velocity_array,      // velocity array (first all x components, then all y components, then z...)
        T *density,             // densities
        int *bc,                // boundary conditions
        T drivenCavityVelocity, // velocity parameters for modification of density distributions
        const int domainCells_x,
        const int domainCells_y,
        const int domainCells_z)
{
    size_t DOMAIN_CELLS = domainCells_x * domainCells_y * domainCells_z;

    // get unique global ID
    size_t blockId = blockIdx.x + (size_t)(blockIdx.y * gridDim.x) + (size_t)(gridDim.x * gridDim.y * blockIdx.z);
    size_t gid = blockId * (size_t)(blockDim.x * blockDim.y * blockDim.z) + (size_t)(threadIdx.z * (blockDim.x * blockDim.y)) + (size_t)(threadIdx.y * blockDim.x) + threadIdx.x;

    if (gid >= DOMAIN_CELLS)
        return;

    //__global T *current_dds = &global_dd[gid];
    T *current_dds = &global_dd[gid];

    // initialize flag field
    int3 pos;
    pos = getCubePosition(gid, domainCells_x, domainCells_y);

    T velocity_x = 0;
    T velocity_y = 0;
    T velocity_z = 0;

    int flag = FLAG_FLUID;

    if(pos.x == 0)
        flag = bc[0];
    else if(pos.x == domainCells_x-1)
        flag = bc[1];

    else if(pos.y == 0)
        flag = bc[2];
    else if(pos.y == domainCells_y-1)
        flag = bc[3];

    else if(pos.z == 0)
        flag = bc[4];
    else if(pos.z == domainCells_z-1)
        flag = bc[5];

//  else if (pos.y == domainCells_y-2)
//      flag = FLAG_VELOCITY_INJECTION;

#if 0
    if (    pos.x == 0 || pos.y == 0 || pos.z == 0 ||
        pos.x == domainCells_x-1 || pos.y == domainCells_y-1 || pos.z == domainCells_z-1
    )
    {
        flag = FLAG_OBSTACLE;
    }
    else
    {
#if 1
        if (pos.y == domainCells_y-2)
            flag = FLAG_VELOCITY_INJECTION;
#endif
#if 0
        if (pos.y == 10)
            flag = FLAG_OBSTACLE;

        if (pos.y == 2)
            velocity_x = 10;
        if (pos.y == 3)
            velocity_x = 10;
        if (pos.y == 4)
            velocity_x = 10;
        if (pos.y == 5)
            velocity_x = 10;
        if (pos.y == 6)
            velocity_x = 10;
#endif
#if 0
        if ((pos.x == domainCells_x/2 || pos.x == domainCells_x-2) && pos.y <= domainCells_y/2)
        {
            flag = FLAG_INTERFACE;
        }
        else if ((pos.y == domainCells_y/2 || pos.y == 1) && pos.x >= domainCells_x/2)
        {
            flag = FLAG_INTERFACE;
        }
        else if ((pos.z == domainCells_z-1 || pos.z == 1) && pos.x >= domainCells_x/2 && pos.y <= domainCells_y/2)
        {
            flag = FLAG_INTERFACE;
        }
        else if (pos.x < domainCells_x/2 || pos.y > domainCells_y/2)
        {
            flag = FLAG_GAS;
        }
#endif
    }
#endif

    // density distributions
    T dd0, dd1, dd2, dd3, dd4, dd5, dd6, dd7, dd8, dd9, dd10, dd11, dd12, dd13, dd14, dd15, dd16, dd17, dd18;

    T dd_param;
    T vela2;
    T vela_velb;
    T rho = 1.0f;

    // compute and store velocity

    vela2 = velocity_x*velocity_x;
    dd_param = rho - (T)(3.0f/2.0f)*(vela2);

    dd0 = eq_dd_a0(velocity_x, vela2, dd_param);
    *current_dds = dd0;     current_dds += DOMAIN_CELLS;
    dd1 = eq_dd_a1(velocity_x, vela2, dd_param);
    *current_dds = dd1;     current_dds += DOMAIN_CELLS;

    vela2 = velocity_y*velocity_y;

    dd2 = eq_dd_a0(velocity_y, vela2, dd_param);
    *current_dds = dd2;     current_dds += DOMAIN_CELLS;
    dd3 = eq_dd_a1(velocity_y, vela2, dd_param);
    *current_dds = dd3;     current_dds += DOMAIN_CELLS;


#define vela_velb_2 vela2
    /***********************
     * DD1
     ***********************/
    vela_velb = velocity_x+velocity_y;
    vela_velb_2 = vela_velb*vela_velb;

    dd4 = eq_dd4(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd4;     current_dds += DOMAIN_CELLS;
    dd5 = eq_dd5(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd5;     current_dds += DOMAIN_CELLS;

    vela_velb = velocity_x-velocity_y;
    vela_velb_2 = vela_velb*vela_velb;

    dd6 = eq_dd4(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd6;     current_dds += DOMAIN_CELLS;
    dd7 = eq_dd5(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd7;     current_dds += DOMAIN_CELLS;

    /***********************
     * DD2
     ***********************/
    vela_velb = velocity_x+velocity_z;
    vela_velb_2 = vela_velb*vela_velb;

    dd8 = eq_dd4(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd8;     current_dds += DOMAIN_CELLS;
    dd9 = eq_dd5(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd9;     current_dds += DOMAIN_CELLS;

    vela_velb = velocity_x-velocity_z;
    vela_velb_2 = vela_velb*vela_velb;

    dd10 = eq_dd4(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd10;        current_dds += DOMAIN_CELLS;
    dd11 = eq_dd5(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd11;        current_dds += DOMAIN_CELLS;

    /***********************
     * DD3
     ***********************/
    vela_velb = velocity_y+velocity_z;
    vela_velb_2 = vela_velb*vela_velb;


    dd12 = eq_dd4(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd12;        current_dds += DOMAIN_CELLS;
    dd13 = eq_dd5(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd13;        current_dds += DOMAIN_CELLS;

    vela_velb = velocity_y-velocity_z;
    vela_velb_2 = vela_velb*vela_velb;

    dd14 = eq_dd4(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd14;        current_dds += DOMAIN_CELLS;
    dd15 = eq_dd5(vela_velb, vela_velb_2, dd_param);
    *current_dds = dd15;        current_dds += DOMAIN_CELLS;


#undef vela_velb_2
    /***********************
     * DD4
     ***********************/
    vela2 = velocity_z*velocity_z;

    dd16 = eq_dd_a0(velocity_z, vela2, dd_param);
    *current_dds = dd16;        current_dds += DOMAIN_CELLS;
    dd17 = eq_dd_a1(velocity_z, vela2, dd_param);
    *current_dds = dd17;        current_dds += DOMAIN_CELLS;

    dd18 = eq_dd18(dd_param);
    *current_dds = dd18;

    // flag
    flags[gid] = flag;

#if STORE_VELOCITY
    // store velocity
    current_dds = &velocity_array[gid];
    *current_dds = velocity_x;  current_dds += DOMAIN_CELLS;
    *current_dds = velocity_y;  current_dds += DOMAIN_CELLS;
    *current_dds = velocity_z;
#endif

#if STORE_DENSITY
    // store density
    density[gid] = rho;
    // density[gid] = flag;
#endif
}

template __global__ void init_kernel<float>(
        float *global_dd,
        int *flags,
        float *velocity_array,
        float *density,
        int *bc,
        float drivenCavityVelocity,
        const int domainCells_x,
        const int domainCells_y,
        const int domainCells_z);
template __global__ void init_kernel<double>(
        double *global_dd,
        int *flags,
        double *velocity_array,
        double *density,
        int *bc,
        double drivenCavityVelocity,
        const int domainCells_x,
        const int domainCells_y,
        const int domainCells_z);
