#include "hip/hip_runtime.h"
/*
 * Copyright
 * 2010 Martin Schreiber
 * 2013 Arash Bakhtiari
 * 2016 Christoph Riesinger, Ayman Saleem
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../common.cuh"
#include "../common.h"

#define GRAVITATION         0

#define CACHED_ACCESS       0

#define USE_SHARED_MEMORY   1

__device__ size_t DOMAIN_WRAP(size_t A, size_t DOMAIN_CELLS, bool isPowTwo)
{
    return ( (int)isPowTwo*(A & (DOMAIN_CELLS-1)) + (int)(!isPowTwo)*(A % DOMAIN_CELLS) );
}

__device__ size_t LOCAL_WORK_GROUP_WRAP(size_t A, size_t LOCAL_WORK_GROUP_SIZE, bool isPowTwo)
{
    return ( (int)isPowTwo * (A & (LOCAL_WORK_GROUP_SIZE-1)) + (int)(!isPowTwo) * (A % LOCAL_WORK_GROUP_SIZE) );
}

template<typename T>
__global__ void lbm_kernel_beta(
        T *global_dd,                 // density distributions
        const int *flag_array,        // flags
        T *velocity,                  // velocities
        T *density,                   // densities
        const T inv_tau,
        const T gravitation_x,
        const T gravitation_y,
        const T gravitation_z,
        const T drivenCavityVelocity, // velocity parameters for modification of density distributions
        const int domainCells_x,
        const int domainCells_y,
        const int domainCells_z,
        const size_t localWorkGroup,
        bool isDomainPowOfTwo,
        bool isLocalPowOfTwo)
{
    //const size_t gid = get_global_id(0);
    const int DOMAIN_CELLS_X = domainCells_x;

#if USE_SHARED_MEMORY
    size_t LOCAL_WORK_GROUP_SIZE = localWorkGroup;
#endif

    size_t DOMAIN_CELLS = DOMAIN_CELLS_X * domainCells_y * domainCells_z;
    size_t DOMAIN_SLICE_CELLS = DOMAIN_CELLS_X * domainCells_y;

    int DELTA_POS_X = 1;
    size_t DELTA_NEG_X = DOMAIN_CELLS - 1;
    int DELTA_POS_Y = DOMAIN_CELLS_X;
    size_t DELTA_NEG_Y = DOMAIN_CELLS - DOMAIN_CELLS_X;
    size_t DELTA_POS_Z = DOMAIN_SLICE_CELLS;
    size_t DELTA_NEG_Z = DOMAIN_CELLS - DOMAIN_SLICE_CELLS;

    // get unique thread id
    size_t blockId = blockIdx.x + (size_t)(blockIdx.y * gridDim.x) + (size_t)(gridDim.x * gridDim.y * blockIdx.z);
    size_t gid = blockId * (size_t)(blockDim.x * blockDim.y * blockDim.z) + (size_t)(threadIdx.z * (blockDim.x * blockDim.y)) + (size_t)(threadIdx.y * blockDim.x) + threadIdx.x;

    if (gid >= DOMAIN_CELLS)
        return;

    // load cell type flag
    const int flag = flag_array[gid];

    /**
     * we use a pointer instead of accessing the array directly
     * first this reduces the number of use registers (according to profiling information)
     * secondly the program runs faster and we can use more threads
     */
    //__global T *current_dds = global_dd;
    T *current_dds = global_dd;

    // velocity
    T velocity_x, velocity_y, velocity_z;

    // density distributions
    T dd0, dd1, dd2, dd3, dd4, dd5, dd6, dd7, dd8, dd9, dd10, dd11, dd12, dd13, dd14, dd15, dd16, dd17, dd18;

    // density
    T rho;

#if !USE_SHARED_MEMORY
    /*
     * dd 0-3: f(1,0,0), f(-1,0,0),  f(0,1,0),  f(0,-1,0)
     */
    dd1 = current_dds[DOMAIN_WRAP(gid + DELTA_POS_X, DOMAIN_CELLS, isDomainPowOfTwo)];  
    current_dds += DOMAIN_CELLS;
    dd0 = current_dds[DOMAIN_WRAP(gid + DELTA_NEG_X, DOMAIN_CELLS, isDomainPowOfTwo)];  
    current_dds += DOMAIN_CELLS;

    rho = dd0;
    velocity_x = dd0;

    /*
     * we have to sum the densities up in a specific order.
     * otherwise it seems that we run into numerical errors for fluids with zero velocity.
     */
    rho += dd1;
    velocity_x -= dd1;

    dd3 = current_dds[DOMAIN_WRAP(gid + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo)];      
    current_dds += DOMAIN_CELLS;
    dd2 = current_dds[DOMAIN_WRAP(gid + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo)];      
    current_dds += DOMAIN_CELLS;

    rho += dd2;
    velocity_y = dd2;

    rho += dd3;
    velocity_y -= dd3;

    /*
     * dd 4-7: f(1,1,0), f(-1,-1,0), f(1,-1,0), f(-1,1,0)
     */
    dd5 = current_dds[DOMAIN_WRAP(gid + (DELTA_POS_X + DELTA_POS_Y), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;
    dd4 = current_dds[DOMAIN_WRAP(gid + (DELTA_NEG_X + DELTA_NEG_Y), DOMAIN_CELLS, isDomainPowOfTwo)];      
    current_dds += DOMAIN_CELLS;

    rho += dd4;
    velocity_x += dd4;
    velocity_y += dd4;

    rho += dd5;
    velocity_x -= dd5;
    velocity_y -= dd5;

    dd7 = current_dds[DOMAIN_WRAP(gid + (DELTA_POS_X + DELTA_NEG_Y), DOMAIN_CELLS, isDomainPowOfTwo)];      
    current_dds += DOMAIN_CELLS;
    dd6 = current_dds[DOMAIN_WRAP(gid + (DELTA_NEG_X + DELTA_POS_Y), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;


    rho += dd6;
    velocity_x += dd6;
    velocity_y -= dd6;

    rho += dd7;
    velocity_x -= dd7;
    velocity_y += dd7;

    /*
     * dd 8-11: f(1,0,1), f(-1,0,-1), f(1,0,-1), f(-1,0,1)
     */
    dd9 = current_dds[DOMAIN_WRAP(gid + (DELTA_POS_X + DELTA_POS_Z), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;
    dd8 = current_dds[DOMAIN_WRAP(gid + (DELTA_NEG_X + DELTA_NEG_Z), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;

    rho += dd8;
    velocity_x += dd8;
    velocity_z = dd8;

    rho += dd9;
    velocity_x -= dd9;
    velocity_z -= dd9;

    dd11 = current_dds[DOMAIN_WRAP(gid + (DELTA_POS_X + DELTA_NEG_Z), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;
    dd10 = current_dds[DOMAIN_WRAP(gid + (DELTA_NEG_X + DELTA_POS_Z), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;

    rho += dd10;
    velocity_x += dd10;
    velocity_z -= dd10;

    rho += dd11;
    velocity_x -= dd11;
    velocity_z += dd11;

    /*
     * dd 12-15: f(0,1,1), f(0,-1,-1), f(0,1,-1), f(0,-1,1)
     */
    dd13 = current_dds[DOMAIN_WRAP(gid + (DELTA_POS_Y + DELTA_POS_Z), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;
    dd12 = current_dds[DOMAIN_WRAP(gid + (DELTA_NEG_Y + DELTA_NEG_Z), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;

    rho += dd12;
    velocity_y += dd12;
    velocity_z += dd12;

    rho += dd13;
    velocity_y -= dd13;
    velocity_z -= dd13;

    dd15 = current_dds[DOMAIN_WRAP(gid + (DELTA_POS_Y + DELTA_NEG_Z), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;
    dd14 = current_dds[DOMAIN_WRAP(gid + (DELTA_NEG_Y + DELTA_POS_Z), DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;

    rho += dd14;
    velocity_y += dd14;
    velocity_z -= dd14;

    rho += dd15;
    velocity_y -= dd15;
    velocity_z += dd15;

    /*
     * dd 16-18: f(0,0,1), f(0,0,-1),  f(0,0,0),  (not used)
     */
    dd17 = current_dds[DOMAIN_WRAP(gid + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;
    dd16 = current_dds[DOMAIN_WRAP(gid + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)];
    current_dds += DOMAIN_CELLS;

    rho += dd16;
    velocity_z += dd16;

    rho += dd17;
    velocity_z -= dd17;

    dd18 = current_dds[gid];
    rho += dd18;

#else
    //__local T dd_buf[12][LOCAL_WORK_GROUP_SIZE];
    // extern __shared__ T dd_buf[];
    SharedMemory<T> smem;
    T* dd_buf = smem.getPointer();

    //const size_t lid = get_local_id(0);
    int lid = threadIdx.x + blockDim.x * threadIdx.y;

#if CACHED_ACCESS
    size_t dd_read_delta_position_2;
    size_t dd_read_delta_position_3;
    size_t dd_read_delta_position_4;
    size_t dd_read_delta_position_5;
    size_t dd_read_delta_position_6;
    size_t dd_read_delta_position_7;
    size_t dd_read_delta_position_8;
    size_t dd_read_delta_position_9;
    size_t dd_read_delta_position_10;
    size_t dd_read_delta_position_11;
    size_t dd_read_delta_position_12;
    size_t dd_read_delta_position_13;
    size_t dd_read_delta_position_14;
    size_t dd_read_delta_position_15;
    size_t dd_read_delta_position_16;
    size_t dd_read_delta_position_17;
    size_t dd_read_delta_position_18;
#endif

    /*
     * We have to handle "misaligned" data with a shift of -1 and +1:
     *
     * As an example, we handle the access to the density distributions with a shift of -1:
     *
     * We use "(( (lid + 1) mod LOCAL_WORK_GROUP_SIZE) + (DOMAIN_CELLS+1)) mod DOMAIN_CELLS" as the reading index
     *          shift to right                              shift back
     *
     * This allows us to read almost everything (except the first thread) aligned. After that the data is stored
     * to shared memory, a sync operation is called and finally the tread can read the originally required data which
     * was previously read by another thread.
     *
     * Every float is stored to a local memory array indexed by the thread id.
     * after the dd's are read from global memory, the local memory is accessed with
     * "(lid + (LOCAL_WORK_GROUP_SIZE-1)) mod LOCAL_WORK_GROUP_SIZE"
     */

    /*
     * pos_x_wrap specifies the position in the local buffer for the dd with the displacement x=+1
     * this is used to force the thread with the largest number to read the dd at the displacement -1 (see gid_pos below)
     * pos_x_wrap and gid_pos have the following values for given local thread ids:
     *  
     * thread_id:   0 1 2 3 4 5 ... 63
     * pos_x_wrap:  1 2 3 4 5 6 ... 0
     * gid_pos:     0 1 2 3 4 5 ... -1   <<< !!!
     */

    int pos_x_wrap = LOCAL_WORK_GROUP_WRAP(lid + 1, LOCAL_WORK_GROUP_SIZE, isLocalPowOfTwo);
    int neg_x_wrap = LOCAL_WORK_GROUP_WRAP(lid + (LOCAL_WORK_GROUP_SIZE - 1), LOCAL_WORK_GROUP_SIZE, isLocalPowOfTwo);

    bool local_to_global = ((LOCAL_WORK_GROUP_SIZE/DOMAIN_CELLS_X) * DOMAIN_CELLS_X == LOCAL_WORK_GROUP_SIZE);
//#if (LOCAL_WORK_GROUP_SIZE/DOMAIN_CELLS_X)*DOMAIN_CELLS_X == LOCAL_WORK_GROUP_SIZE
#if local_to_global 
    /*
     * handle domain x-sizes specially if LOCAL_WORK_GROUP_SIZE is a multiple of DOMAIN_CELLS_X
     * in this case, we dont have to read unaligned data!!!
     */
    int read_delta_neg_x = gid;
    int read_delta_pos_x = gid;
#else
    /*
     * cache variables for speedup
     */
    int read_delta_neg_x = DOMAIN_WRAP(gid - lid + pos_x_wrap + DELTA_NEG_X, DOMAIN_CELLS, isDomainPowOfTwo);
    int read_delta_pos_x = DOMAIN_WRAP(gid - lid + neg_x_wrap + DELTA_POS_X, DOMAIN_CELLS, isDomainPowOfTwo);
#endif

    /*
     * +++++++++++
     * +++ DD0 +++
     * +++++++++++
     *
     * dd0: f(1,0,0), f(-1,0,0),  f(0,1,0),  f(0,-1,0)
     * negative displacement
     * preload to alignment buffer
     */

    /*
     * read negative distribution vector (-1,0,0) from relative cell (-1,0,0) and store it to positive distribution vector (1,0,0)
     *
     * in the alpha kernel, the density distribution values have been stored to the oppisite density distribution storage
     * to avoid the propagation step
     */

    /*
     * pointer to current dd buf entry with index lid
     */
    //__local T *dd_buf_lid = &dd_buf[1][lid];
    //extern __shared__    T dd_buf_lid[1];
    //T *dd_buf_lid = &dd_buf[1][lid];
    T *dd_buf_lid = &dd_buf[1 * LOCAL_WORK_GROUP_SIZE + lid];

#define dd_read_delta_position_0    read_delta_neg_x
#define dd_read_delta_position_1    read_delta_pos_x

    *dd_buf_lid = current_dds[dd_read_delta_position_1];        current_dds += DOMAIN_CELLS;    dd_buf_lid -= LOCAL_WORK_GROUP_SIZE;
    *dd_buf_lid = current_dds[dd_read_delta_position_0];        current_dds += DOMAIN_CELLS;    dd_buf_lid += 5*LOCAL_WORK_GROUP_SIZE;
#if CACHED_ACCESS
    dd_read_delta_position_3 = DOMAIN_WRAP(gid + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_3    DOMAIN_WRAP(gid + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    dd3 = current_dds[dd_read_delta_position_3];        current_dds += DOMAIN_CELLS;
    rho = dd3;
    velocity_y = -dd3;

#if CACHED_ACCESS
    dd_read_delta_position_2 = DOMAIN_WRAP(gid + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_2    DOMAIN_WRAP(gid + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    dd2 = current_dds[dd_read_delta_position_2];        current_dds += DOMAIN_CELLS;
    rho += dd2;
    velocity_y += dd2;

    // DD0 STUFF
    //barrier(CLK_LOCAL_MEM_FENCE); 
    __syncthreads();

    //dd0 = dd_buf[0][neg_x_wrap];
    dd0 = dd_buf[0 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap];
    rho += dd0;
    velocity_x = dd0;

    //dd1 = dd_buf[1][pos_x_wrap];
    dd1 = dd_buf[1 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap];
    rho += dd1;
    velocity_x -= dd1;


    /* +++++++++++
     * +++ DD1 +++
     * +++++++++++
     *
     * dd1: f(1,1,0), f(-1,-1,0), f(1,-1,0), f(-1,1,0)
     */
#if CACHED_ACCESS
    dd_read_delta_position_5 = DOMAIN_WRAP(read_delta_pos_x + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_5    DOMAIN_WRAP(read_delta_pos_x + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    *dd_buf_lid = current_dds[dd_read_delta_position_5];    current_dds += DOMAIN_CELLS;    dd_buf_lid -= LOCAL_WORK_GROUP_SIZE;

#if CACHED_ACCESS
    dd_read_delta_position_4 = DOMAIN_WRAP(read_delta_neg_x + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_4    DOMAIN_WRAP(read_delta_neg_x + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    *dd_buf_lid = current_dds[dd_read_delta_position_4];    current_dds += DOMAIN_CELLS;    dd_buf_lid += 3*LOCAL_WORK_GROUP_SIZE;

#if CACHED_ACCESS
    dd_read_delta_position_7 = DOMAIN_WRAP(read_delta_pos_x + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_7    DOMAIN_WRAP(read_delta_pos_x + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    *dd_buf_lid = current_dds[dd_read_delta_position_7];    current_dds += DOMAIN_CELLS;    dd_buf_lid -= LOCAL_WORK_GROUP_SIZE;

#if CACHED_ACCESS
    dd_read_delta_position_6 = DOMAIN_WRAP(read_delta_neg_x + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_6    DOMAIN_WRAP(read_delta_neg_x + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    *dd_buf_lid = current_dds[dd_read_delta_position_6];    current_dds += DOMAIN_CELLS;    dd_buf_lid += 3*LOCAL_WORK_GROUP_SIZE;

    //barrier(CLK_LOCAL_MEM_FENCE);
    __syncthreads();

    //dd4 = dd_buf[4][neg_x_wrap];
    dd4 = dd_buf[4 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap];
    rho += dd4;
    velocity_x += dd4;
    velocity_y += dd4;

    //dd5 = dd_buf[5][pos_x_wrap];
    dd5 = dd_buf[5 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap];
    rho += dd5;
    velocity_x -= dd5;
    velocity_y -= dd5;


    //dd6 = dd_buf[6][neg_x_wrap];
    dd6 = dd_buf[6 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap];
    rho += dd6;
    velocity_x += dd6;
    velocity_y -= dd6;

    //dd7 = dd_buf[7][pos_x_wrap];
    dd7 = dd_buf[7 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap];
    rho += dd7;
    velocity_x -= dd7;
    velocity_y += dd7;


    /* +++++++++++
     * +++ DD2 +++
     * +++++++++++
     *
     * dd2: f(1,0,1), f(-1,0,-1), f(1,0,-1), f(-1,0,1)
     */

#if CACHED_ACCESS
    dd_read_delta_position_9 = DOMAIN_WRAP(read_delta_pos_x + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_9    DOMAIN_WRAP(read_delta_pos_x + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    *dd_buf_lid = current_dds[dd_read_delta_position_9];    current_dds += DOMAIN_CELLS;    dd_buf_lid -= LOCAL_WORK_GROUP_SIZE;

#if CACHED_ACCESS
    dd_read_delta_position_8 = DOMAIN_WRAP(read_delta_neg_x + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_8    DOMAIN_WRAP(read_delta_neg_x + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    *dd_buf_lid = current_dds[dd_read_delta_position_8];    current_dds += DOMAIN_CELLS;    dd_buf_lid += 3*LOCAL_WORK_GROUP_SIZE;

#if CACHED_ACCESS
    dd_read_delta_position_11 = DOMAIN_WRAP(read_delta_pos_x + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_11   DOMAIN_WRAP(read_delta_pos_x + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    *dd_buf_lid = current_dds[dd_read_delta_position_11];   current_dds += DOMAIN_CELLS;    dd_buf_lid -= LOCAL_WORK_GROUP_SIZE;

#if CACHED_ACCESS
    dd_read_delta_position_10 = DOMAIN_WRAP(read_delta_neg_x + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_10   DOMAIN_WRAP(read_delta_neg_x + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    *dd_buf_lid = current_dds[dd_read_delta_position_10];   current_dds += DOMAIN_CELLS;


    //barrier(CLK_LOCAL_MEM_FENCE);
    __syncthreads();

    //dd8 = dd_buf[8][neg_x_wrap];
    dd8 = dd_buf[8 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap];
    rho += dd8;
    velocity_x += dd8;
    velocity_z = dd8;

    //dd9 = dd_buf[9][pos_x_wrap];
    dd9 = dd_buf[9 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap];
    rho += dd9;
    velocity_x -= dd9;
    velocity_z -= dd9;

    //dd10 = dd_buf[10][neg_x_wrap];
    dd10 = dd_buf[10 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap];
    rho += dd10;
    velocity_x += dd10;
    velocity_z -= dd10;

    //dd11 = dd_buf[11][pos_x_wrap];
    dd11 = dd_buf[11 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap];
    rho += dd11;
    velocity_x -= dd11;
    velocity_z += dd11;


    /*
     * +++++++++++
     * +++ DD3 +++
     * +++++++++++
     *
     * dd3: f(0,1,1), f(0,-1,-1), f(0,1,-1), f(0,-1,1)
     */

#if CACHED_ACCESS
    dd_read_delta_position_13 = DOMAIN_WRAP(gid + DELTA_POS_Y + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_13   DOMAIN_WRAP(gid + DELTA_POS_Y + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    dd13 = current_dds[dd_read_delta_position_13];  current_dds += DOMAIN_CELLS;
    rho += dd13;
    velocity_y -= dd13;
    velocity_z -= dd13;

#if CACHED_ACCESS
    dd_read_delta_position_12 = DOMAIN_WRAP(gid + DELTA_NEG_Y + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_12   DOMAIN_WRAP(gid + DELTA_NEG_Y + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    dd12 = current_dds[dd_read_delta_position_12];  current_dds += DOMAIN_CELLS;
    rho += dd12;
    velocity_y += dd12;
    velocity_z += dd12;

#if CACHED_ACCESS
    dd_read_delta_position_15 = DOMAIN_WRAP(gid + DELTA_POS_Y + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_15   DOMAIN_WRAP(gid + DELTA_POS_Y + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    dd15 = current_dds[dd_read_delta_position_15];  current_dds += DOMAIN_CELLS;
    rho += dd15;
    velocity_y -= dd15;
    velocity_z += dd15;

#if CACHED_ACCESS
    dd_read_delta_position_14 = DOMAIN_WRAP(gid + DELTA_NEG_Y + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_14   DOMAIN_WRAP(gid + DELTA_NEG_Y + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    dd14 = current_dds[dd_read_delta_position_14];  current_dds += DOMAIN_CELLS;
    rho += dd14;
    velocity_y += dd14;
    velocity_z -= dd14;


    /*
     * +++++++++++
     * +++ DD4 +++
     * +++++++++++
     *
     * dd4: f(0,0,1), f(0,0,-1),  f(0,0,0),  (not used)
     */
#if CACHED_ACCESS
    dd_read_delta_position_17 = DOMAIN_WRAP(gid + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_17   DOMAIN_WRAP(gid + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    dd17 = current_dds[dd_read_delta_position_17];  current_dds += DOMAIN_CELLS;
    rho += dd17;
    velocity_z -= dd17;

#if CACHED_ACCESS
    dd_read_delta_position_16 = DOMAIN_WRAP(gid + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo);
#else
    #define dd_read_delta_position_16   DOMAIN_WRAP(gid + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)
#endif
    dd16 = current_dds[dd_read_delta_position_16];  current_dds += DOMAIN_CELLS;
    rho += dd16;
    velocity_z += dd16;

    dd18 = current_dds[gid];
    rho += dd18;

#endif // !USE_SHARED_MEMORY

    T vel2;     // vel*vel
    T vela2;

#define vela_velb   vel2
#define vela_velb_2 vela2

#define dd_param    rho
//  T dd_param; // modified rho as temporary variable
    switch(flag)
    {
        case FLAG_FLUID:    // this is the whole collision operator
            vel2 = velocity_x*velocity_x + velocity_y*velocity_y + velocity_z*velocity_z;
            dd_param = rho - (T)(3.0f/2.0f)*(vel2);

            vela2 = velocity_x*velocity_x;
            dd0 += inv_tau*(eq_dd_a0(velocity_x, vela2, dd_param) - dd0);
            dd1 += inv_tau*(eq_dd_a1(velocity_x, vela2, dd_param) - dd1);

            vela2 = velocity_y*velocity_y;
            dd2 += inv_tau*(eq_dd_a0(velocity_y, vela2, dd_param) - dd2);
            dd3 += inv_tau*(eq_dd_a1(velocity_y, vela2, dd_param) - dd3);


            /***********************
             * DD1
             ***********************/
            vela_velb = velocity_x+velocity_y;
            vela_velb_2 = vela_velb*vela_velb;

            dd4 += inv_tau*(eq_dd4(vela_velb, vela_velb_2, dd_param) - dd4);
            dd5 += inv_tau*(eq_dd5(vela_velb, vela_velb_2, dd_param) - dd5);

            vela_velb = velocity_x-velocity_y;
            vela_velb_2 = vela_velb*vela_velb;

            dd6 += inv_tau*(eq_dd4(vela_velb, vela_velb_2, dd_param) - dd6);
            dd7 += inv_tau*(eq_dd5(vela_velb, vela_velb_2, dd_param) - dd7);

            /***********************
             * DD2
             ***********************/
            vela_velb = velocity_x+velocity_z;
            vela_velb_2 = vela_velb*vela_velb;
            dd8 += inv_tau*(eq_dd4(vela_velb, vela_velb_2, dd_param) - dd8);
            dd9 += inv_tau*(eq_dd5(vela_velb, vela_velb_2, dd_param) - dd9);

            vela_velb = velocity_x-velocity_z;
            vela_velb_2 = vela_velb*vela_velb;
            dd10 += inv_tau*(eq_dd4(vela_velb, vela_velb_2, dd_param) - dd10);
            dd11 += inv_tau*(eq_dd5(vela_velb, vela_velb_2, dd_param) - dd11);

            /***********************
             * DD3
             ***********************/
            vela_velb = velocity_y+velocity_z;
            vela_velb_2 = vela_velb*vela_velb;
            dd12 += inv_tau*(eq_dd4(vela_velb, vela_velb_2, dd_param) - dd12);
            dd13 += inv_tau*(eq_dd5(vela_velb, vela_velb_2, dd_param) - dd13);

            vela_velb = velocity_y-velocity_z;
            vela_velb_2 = vela_velb*vela_velb;
            dd14 += inv_tau*(eq_dd4(vela_velb, vela_velb_2, dd_param) - dd14);
            dd15 += inv_tau*(eq_dd5(vela_velb, vela_velb_2, dd_param) - dd15);

            /***********************
             * DD4
             ***********************/
            vela2 = velocity_z*velocity_z;
            dd16 += inv_tau*(eq_dd_a0(velocity_z, vela2, dd_param) - dd16);
            dd17 += inv_tau*(eq_dd_a1(velocity_z, vela2, dd_param) - dd17);

            dd18 += inv_tau*(eq_dd18(dd_param) - dd18);
            break;

        case FLAG_OBSTACLE: // in case of an obstacle, we bounce back the values
            // set to zero velocity and no fluid density
#if STORE_VELOCITY
            velocity_x = 0.0f;
            velocity_y = 0.0f;
            velocity_z = 0.0f;
#endif

            // use simple bounce back
            vela2 = dd1;    dd1 = dd0;      dd0 = vela2;
            vela2 = dd3;    dd3 = dd2;      dd2 = vela2;
            vela2 = dd5;    dd5 = dd4;      dd4 = vela2;
            vela2 = dd7;    dd7 = dd6;      dd6 = vela2;
            vela2 = dd9;    dd9 = dd8;      dd8 = vela2;
            vela2 = dd11;   dd11 = dd10;    dd10 = vela2;
            vela2 = dd13;   dd13 = dd12;    dd12 = vela2;
            vela2 = dd15;   dd15 = dd14;    dd14 = vela2;
            vela2 = dd17;   dd17 = dd16;    dd16 = vela2;

            break;

        case FLAG_VELOCITY_INJECTION:   // this flag specifies the injection area of the fluid
            velocity_x = drivenCavityVelocity;
            velocity_y = 0;
            velocity_z = 0;

            rho = 1.0f;

            vel2 = velocity_x*velocity_x + velocity_y*velocity_y + velocity_z*velocity_z;
            dd_param = rho - (T)(3.0f/2.0f)*(vel2);

            /***********************
             * DD0
             ***********************/
            vela2 = velocity_x*velocity_x;
            dd0 = eq_dd_a0(velocity_x, vela2, dd_param);
            dd1 = eq_dd_a1(velocity_x, vela2, dd_param);

            vela2 = velocity_y*velocity_y;
            dd2 = eq_dd_a0(velocity_y, vela2, dd_param);
            dd3 = eq_dd_a1(velocity_y, vela2, dd_param);

            /***********************
             * DD1
             ***********************/
            vela_velb = velocity_x+velocity_y;
            vela_velb_2 = vela_velb*vela_velb;

            dd4 = eq_dd4(vela_velb, vela_velb_2, dd_param);
            dd5 = eq_dd5(vela_velb, vela_velb_2, dd_param);

            vela_velb = velocity_x-velocity_y;
            vela_velb_2 = vela_velb*vela_velb;
            dd6 = eq_dd4(vela_velb, vela_velb_2, dd_param);
            dd7 = eq_dd5(vela_velb, vela_velb_2, dd_param);

            /***********************
             * DD2
             ***********************/
            vela_velb = velocity_x+velocity_z;
            vela_velb_2 = vela_velb*vela_velb;

            dd8 = eq_dd4(vela_velb, vela_velb_2, dd_param);
            dd9 = eq_dd5(vela_velb, vela_velb_2, dd_param);

            vela_velb = velocity_x-velocity_z;
            vela_velb_2 = vela_velb*vela_velb;
            dd10 = eq_dd4(vela_velb, vela_velb_2, dd_param);
            dd11 = eq_dd5(vela_velb, vela_velb_2, dd_param);

            /***********************
             * DD3
             ***********************/
            vela_velb = velocity_y+velocity_z;
            vela_velb_2 = vela_velb*vela_velb;

            dd12 = eq_dd4(vela_velb, vela_velb_2, dd_param);
            dd13 = eq_dd5(vela_velb, vela_velb_2, dd_param);

            vela_velb = velocity_y-velocity_z;
            vela_velb_2 = vela_velb*vela_velb;
            dd14 = eq_dd4(vela_velb, vela_velb_2, dd_param);
            dd15 = eq_dd5(vela_velb, vela_velb_2, dd_param);

            /***********************
             * DD4
             ***********************/
            vela2 = velocity_z*velocity_z;
            dd16 = eq_dd_a0(velocity_z, vela2, dd_param);
            dd17 = eq_dd_a1(velocity_z, vela2, dd_param);

            dd18 = eq_dd18(dd_param);
            break;
        case ( FLAG_GHOST_LAYER):
            break;
    }

#if GRAVITATION
    /*
    velocity_x += 10.0*gravitation_x;
    velocity_y += 10.0*gravitation_y;
    velocity_z += 10.0*gravitation_z;
    */

#define tmp vela2
    if (flag != FLAG_OBSTACLE)
    {
        tmp = gravitation_x*(T)(1.0f/18.0f)*rho;
        dd0 += tmp;
        dd1 -= tmp;
        tmp = gravitation_y*(T)(-1.0f/18.0f)*rho;
        dd2 += tmp;
        dd3 -= tmp;

        tmp = (gravitation_x - gravitation_y)*(T)(1.0f/36.0f)*rho;
        dd4 += tmp;
        dd5 -= tmp;
        tmp = (gravitation_x + gravitation_y)*(T)(1.0f/36.0f)*rho;
        dd6 += tmp;
        dd7 -= tmp;

        tmp = (gravitation_x + gravitation_z)*(T)(1.0f/36.0f)*rho;
        dd8 += tmp;
        dd9 -= tmp;
        tmp = (gravitation_x - gravitation_z)*(T)(1.0f/36.0f)*rho;
        dd10 += tmp;
        dd11 -= tmp;

        tmp = (gravitation_z - gravitation_y)*(T)(1.0f/36.0f)*rho;
        dd12 += tmp;
        dd13 -= tmp;
        tmp = (gravitation_z + gravitation_y)*(T)(-1.0f/36.0f)*rho;
        dd14 += tmp;
        dd15 -= tmp;

        tmp = gravitation_z*(T)(1.0f/18.0f)*rho;
        dd16 += tmp;
        dd17 -= tmp;
    }
#undef tmp

#endif
    // gain little speedup
    //barrier(CLK_LOCAL_MEM_FENCE);
    __syncthreads();

    current_dds = global_dd;

#if USE_SHARED_MEMORY
    //dd_buf_lid = &dd_buf[0][lid];
    dd_buf_lid = &dd_buf[0 * LOCAL_WORK_GROUP_SIZE + lid];

    /* f(1,0,0), f(-1,0,0),  f(0,1,0),  f(0,-1,0) */
    //dd_buf[0][pos_x_wrap] = dd0;
    //dd_buf[1][neg_x_wrap] = dd1;
    dd_buf[0 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap] = dd0;
      dd_buf[1 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap] = dd1;
    //barrier(CLK_LOCAL_MEM_FENCE);
    __syncthreads();

    current_dds[dd_read_delta_position_1] = *dd_buf_lid;    current_dds += DOMAIN_CELLS;    dd_buf_lid += LOCAL_WORK_GROUP_SIZE;
    current_dds[dd_read_delta_position_0] = *dd_buf_lid;    current_dds += DOMAIN_CELLS;    dd_buf_lid += 3*LOCAL_WORK_GROUP_SIZE;
    current_dds[dd_read_delta_position_3] = dd2;        current_dds += DOMAIN_CELLS;
    current_dds[dd_read_delta_position_2] = dd3;        current_dds += DOMAIN_CELLS;

    /* f(1,1,0), f(-1,-1,0), f(1,-1,0), f(-1,1,0) */
//    dd_buf[4][pos_x_wrap] = dd4;
//    dd_buf[5][neg_x_wrap] = dd5;
//    dd_buf[6][pos_x_wrap] = dd6;
//    dd_buf[7][neg_x_wrap] = dd7;
    dd_buf[4 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap] = dd4;
    dd_buf[5 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap] = dd5;
    dd_buf[6 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap] = dd6;
    dd_buf[7 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap] = dd7;
    //barrier(CLK_LOCAL_MEM_FENCE);
    __syncthreads();

    current_dds[dd_read_delta_position_5] = *dd_buf_lid;    current_dds += DOMAIN_CELLS;    dd_buf_lid += LOCAL_WORK_GROUP_SIZE;
    current_dds[dd_read_delta_position_4] = *dd_buf_lid;    current_dds += DOMAIN_CELLS;    dd_buf_lid += LOCAL_WORK_GROUP_SIZE;
    current_dds[dd_read_delta_position_7] = *dd_buf_lid;    current_dds += DOMAIN_CELLS;    dd_buf_lid += LOCAL_WORK_GROUP_SIZE;
    current_dds[dd_read_delta_position_6] = *dd_buf_lid;    current_dds += DOMAIN_CELLS;    dd_buf_lid += LOCAL_WORK_GROUP_SIZE;

    /* f(1,0,1), f(-1,0,-1), f(1,0,-1), f(-1,0,1) */
//    dd_buf[8][pos_x_wrap] = dd8;
//    dd_buf[9][neg_x_wrap] = dd9;
//    dd_buf[10][pos_x_wrap] = dd10;
//    dd_buf[11][neg_x_wrap] = dd11;
    dd_buf[8 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap] = dd8;
    dd_buf[9 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap] = dd9;
    dd_buf[10 * LOCAL_WORK_GROUP_SIZE + pos_x_wrap] = dd10;
    dd_buf[11 * LOCAL_WORK_GROUP_SIZE + neg_x_wrap] = dd11;
    //barrier(CLK_LOCAL_MEM_FENCE);
    __syncthreads();

    current_dds[dd_read_delta_position_9] = *dd_buf_lid;    current_dds += DOMAIN_CELLS;    dd_buf_lid += LOCAL_WORK_GROUP_SIZE;
    current_dds[dd_read_delta_position_8] = *dd_buf_lid;    current_dds += DOMAIN_CELLS;    dd_buf_lid += LOCAL_WORK_GROUP_SIZE;
    current_dds[dd_read_delta_position_11] = *dd_buf_lid;   current_dds += DOMAIN_CELLS;    dd_buf_lid += LOCAL_WORK_GROUP_SIZE;
    current_dds[dd_read_delta_position_10] = *dd_buf_lid;   current_dds += DOMAIN_CELLS;

    /* f(0,1,1), f(0,-1,-1), f(0,1,-1), f(0,-1,1) */
    current_dds[dd_read_delta_position_13] = dd12;  current_dds += DOMAIN_CELLS;
    current_dds[dd_read_delta_position_12] = dd13;  current_dds += DOMAIN_CELLS;
    current_dds[dd_read_delta_position_15] = dd14;  current_dds += DOMAIN_CELLS;
    current_dds[dd_read_delta_position_14] = dd15;  current_dds += DOMAIN_CELLS;

    /* f(0,0,1), f(0,0,-1),  f(0,0,0) */
    current_dds[dd_read_delta_position_17] = dd16;  current_dds += DOMAIN_CELLS;
    current_dds[dd_read_delta_position_16] = dd17;  current_dds += DOMAIN_CELLS;
    current_dds[gid] = dd18;
#else

    /* f(1,0,0), f(-1,0,0),  f(0,1,0),  f(0,-1,0) */
    current_dds[DOMAIN_WRAP(gid + DELTA_POS_X, DOMAIN_CELLS, isDomainPowOfTwo)] = dd0;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_NEG_X, DOMAIN_CELLS, isDomainPowOfTwo)] = dd1;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo)] = dd2;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo)] = dd3;
    current_dds += DOMAIN_CELLS;

    /* f(1,1,0), f(-1,-1,0), f(1,-1,0), f(-1,1,0) */
    current_dds[DOMAIN_WRAP(gid + DELTA_POS_X + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo)] = dd4;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_NEG_X + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo)] = dd5;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_POS_X + DELTA_NEG_Y, DOMAIN_CELLS, isDomainPowOfTwo)] = dd6;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_NEG_X + DELTA_POS_Y, DOMAIN_CELLS, isDomainPowOfTwo)] = dd7;
    current_dds += DOMAIN_CELLS;

    /* f(1,0,1), f(-1,0,-1), f(1,0,-1), f(-1,0,1) */
    current_dds[DOMAIN_WRAP(gid + DELTA_POS_X + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd8;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_NEG_X + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd9;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_POS_X + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd10;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_NEG_X + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd11;
    current_dds += DOMAIN_CELLS;

    /* f(0,1,1), f(0,-1,-1), f(0,1,-1), f(0,-1,1) */
    current_dds[DOMAIN_WRAP(gid + DELTA_POS_Y + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd12;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_NEG_Y + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd13;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_POS_Y + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd14;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_NEG_Y + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd15;
    current_dds += DOMAIN_CELLS;

    /* f(0,0,1), f(0,0,-1),  f(0,0,0) */
    current_dds[DOMAIN_WRAP(gid + DELTA_POS_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd16;
    current_dds += DOMAIN_CELLS;
    current_dds[DOMAIN_WRAP(gid + DELTA_NEG_Z, DOMAIN_CELLS, isDomainPowOfTwo)] = dd17;
    current_dds += DOMAIN_CELLS;
    current_dds[gid] = dd18;
#endif

    if ( flag == FLAG_GHOST_LAYER)
        return;

#if STORE_VELOCITY
    // store velocity
    current_dds = &velocity[gid];
    *current_dds = velocity_x;  current_dds += DOMAIN_CELLS;
    *current_dds = velocity_y;  current_dds += DOMAIN_CELLS;
    *current_dds = velocity_z;
#endif

#if STORE_DENSITY
    // store density (not necessary)
    density[gid] = rho;
#endif
}

template __global__ void lbm_kernel_beta<float>(
        float *global_dd,
        const int *flag_array,
        float *velocity,
        float *density,
        const float inv_tau,
        const float gravitation_x,
        const float gravitation_y,
        const float gravitation_z,
        const float drivenCavityVelocity,
        const int domainCells_x,
        const int domainCells_y,
        const int domainCells_z,
        const size_t localWorkGroup,
        bool isDomainPowOfTwo,
        bool isLocalPowOfTwo);
template __global__ void lbm_kernel_beta<double>(
        double *global_dd,
        const int *flag_array,
        double *velocity,
        double *density,
        const double inv_tau,
        const double gravitation_x,
        const double gravitation_y,
        const double gravitation_z,
        const double drivenCavityVelocity,
        const int domainCells_x,
        const int domainCells_y,
        const int domainCells_z,
        const size_t localWorkGroup,
        bool isDomainPowOfTwo,
        bool isLocalPowOfTwo);
