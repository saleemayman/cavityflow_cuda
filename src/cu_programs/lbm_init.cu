#include "hip/hip_runtime.h"
#include "lbm_header.h"

/**
 * return 3D position
 * \input linear_position	linear position in 3D cube (ordering: X,Y,Z)
 * \return	Vector with 3D position in cube
 */
__device__ inline T4 getCubePosition(int linear_position)
{
	T4 pos;

	// TODO: use AND operation to speed up
	// (but this function is only used during initialization)
	pos.x = (T)((int)linear_position % (int)DOMAIN_CELLS_X);
	linear_position /= DOMAIN_CELLS_X;

	pos.y = (T)((int)linear_position % (int)DOMAIN_CELLS_Y);
	linear_position /= DOMAIN_CELLS_Y;

	pos.z = linear_position;// % CUBE_SIZE_Z;
	return pos;
}


/**
 * INIT KERNEL
 *
 * \param dd	density distributions
 * \param flags	flags of cells
 */
extern "C" __global__ void init_kernel(
		T 	*global_dd,	// density distributions
		int *flags,	// flags
		T 	*velocity_array,	// velocity array (first all x components, then all y components, then z...)
		T 	*density,	// densities
		int *bc, 		///< boundary conditions
		T 	drivenCavityVelocity			// velocity parameters for modification of density distributions
		)
{
	//const size_t gid = get_global_id(0);
	// const size_t gid = threadIdx.x + blockDim.x * blockIdx.x;
	const size_t idx_x = threadIdx.x + blockDim.x * blockIdx.x;
	const size_t idx_y = threadIdx.y + blockDim.y * blockIdx.y;
	const size_t idx_z = threadIdx.z + blockDim.z * blockIdx.z;

	const size_t idx_xy = idx_y * (blockDim.x * gridDim.x) + idx_x;
	const size_t gid = idx_z * (blockDim.x * gridDim.x + blockDim.y * gridDim.y) + idx_xy;

	if (gid >= GLOBAL_WORK_GROUP_SIZE)
		return;

	//__global T *current_dds = &global_dd[gid];
	T *current_dds = &global_dd[gid];

	// initialize flag field
	T4 pos = getCubePosition(gid);
	pos.w = 0;

	T velocity_x = 0;
	T velocity_y = 0;
	T velocity_z = 0;

	int flag = FLAG_FLUID;

	if( pos.x == 0)
		flag = bc[0];
	else if( pos.x == DOMAIN_CELLS_X-1 )
		flag = bc[1];

	else if( pos.y == 0)
		flag = bc[2];
	else if( pos.y == DOMAIN_CELLS_Y-1 )
		flag = bc[3];

	else if( pos.z == 0)
		flag = bc[4];
	else if( pos.z == DOMAIN_CELLS_Z-1 )
		flag = bc[5];

//	else if (pos.y == DOMAIN_CELLS_Y-2)
//		flag = FLAG_VELOCITY_INJECTION;

#if 0
	if (	pos.x == 0 || pos.y == 0 || pos.z == 0 ||
		pos.x == DOMAIN_CELLS_X-1 || pos.y == DOMAIN_CELLS_Y-1 || pos.z == DOMAIN_CELLS_Z-1
	)
	{
		flag = FLAG_OBSTACLE;
	}
	else
	{
#if 1
		if (pos.y == DOMAIN_CELLS_Y-2)
			flag = FLAG_VELOCITY_INJECTION;
#endif
#if 0
		if (pos.y == 10)
			flag = FLAG_OBSTACLE;

		if (pos.y == 2)
			velocity_x = 10;
		if (pos.y == 3)
			velocity_x = 10;
		if (pos.y == 4)
			velocity_x = 10;
		if (pos.y == 5)
			velocity_x = 10;
		if (pos.y == 6)
			velocity_x = 10;
#endif
#if 0
		if ((pos.x == DOMAIN_CELLS_X/2 || pos.x == DOMAIN_CELLS_X-2) && pos.y <= DOMAIN_CELLS_Y/2)
		{
			flag = FLAG_INTERFACE;
		}
		else if ((pos.y == DOMAIN_CELLS_Y/2 || pos.y == 1) && pos.x >= DOMAIN_CELLS_X/2)
		{
			flag = FLAG_INTERFACE;
		}
		else if ((pos.z == DOMAIN_CELLS_Z-1 || pos.z == 1) && pos.x >= DOMAIN_CELLS_X/2 && pos.y <= DOMAIN_CELLS_Y/2)
		{
			flag = FLAG_INTERFACE;
		}
		else if (pos.x < DOMAIN_CELLS_X/2 || pos.y > DOMAIN_CELLS_Y/2)
		{
			flag = FLAG_GAS;
		}
#endif
	}
#endif

	// density distributions
	T dd0, dd1, dd2, dd3, dd4, dd5, dd6, dd7, dd8, dd9, dd10, dd11, dd12, dd13, dd14, dd15, dd16, dd17, dd18;

	T dd_param;
	T vela2;
	T vela_velb;
	T rho = 1.0f;

	// compute and store velocity

	vela2 = velocity_x*velocity_x;
	dd_param = rho - (T)(3.0f/2.0f)*(vela2);

	dd0 = eq_dd_a0(velocity_x, vela2, dd_param);
	*current_dds = dd0;		current_dds += DOMAIN_CELLS;
	dd1 = eq_dd_a1(velocity_x, vela2, dd_param);
	*current_dds = dd1;		current_dds += DOMAIN_CELLS;

	vela2 = velocity_y*velocity_y;

	dd2 = eq_dd_a0(velocity_y, vela2, dd_param);
	*current_dds = dd2;		current_dds += DOMAIN_CELLS;
	dd3 = eq_dd_a1(velocity_y, vela2, dd_param);
	*current_dds = dd3;		current_dds += DOMAIN_CELLS;


#define vela_velb_2	vela2
	/***********************
	 * DD1
	 ***********************/
	vela_velb = velocity_x+velocity_y;
	vela_velb_2 = vela_velb*vela_velb;

	dd4 = eq_dd4(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd4;		current_dds += DOMAIN_CELLS;
	dd5 = eq_dd5(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd5;		current_dds += DOMAIN_CELLS;

	vela_velb = velocity_x-velocity_y;
	vela_velb_2 = vela_velb*vela_velb;

	dd6 = eq_dd4(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd6;		current_dds += DOMAIN_CELLS;
	dd7 = eq_dd5(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd7;		current_dds += DOMAIN_CELLS;

	/***********************
	 * DD2
	 ***********************/
	vela_velb = velocity_x+velocity_z;
	vela_velb_2 = vela_velb*vela_velb;

	dd8 = eq_dd4(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd8;		current_dds += DOMAIN_CELLS;
	dd9 = eq_dd5(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd9;		current_dds += DOMAIN_CELLS;

	vela_velb = velocity_x-velocity_z;
	vela_velb_2 = vela_velb*vela_velb;

	dd10 = eq_dd4(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd10;		current_dds += DOMAIN_CELLS;
	dd11 = eq_dd5(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd11;		current_dds += DOMAIN_CELLS;

	/***********************
	 * DD3
	 ***********************/
	vela_velb = velocity_y+velocity_z;
	vela_velb_2 = vela_velb*vela_velb;


	dd12 = eq_dd4(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd12;		current_dds += DOMAIN_CELLS;
	dd13 = eq_dd5(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd13;		current_dds += DOMAIN_CELLS;

	vela_velb = velocity_y-velocity_z;
	vela_velb_2 = vela_velb*vela_velb;

	dd14 = eq_dd4(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd14;		current_dds += DOMAIN_CELLS;
	dd15 = eq_dd5(vela_velb, vela_velb_2, dd_param);
	*current_dds = dd15;		current_dds += DOMAIN_CELLS;


#undef vela_velb_2
	/***********************
	 * DD4
	 ***********************/
	vela2 = velocity_z*velocity_z;

	dd16 = eq_dd_a0(velocity_z, vela2, dd_param);
	*current_dds = dd16;		current_dds += DOMAIN_CELLS;
	dd17 = eq_dd_a1(velocity_z, vela2, dd_param);
	*current_dds = dd17;		current_dds += DOMAIN_CELLS;

	dd18 = eq_dd18(dd_param);
	*current_dds = dd18;

	// flag
	flags[gid] = flag;

// #if STORE_VELOCITY
#if 1
	// store velocity
	current_dds = &velocity_array[gid];
	*current_dds = velocity_x;	current_dds += DOMAIN_CELLS;
	*current_dds = velocity_y;	current_dds += DOMAIN_CELLS;
	*current_dds = velocity_z;
#endif

#if STORE_DENSITY
// #if 1
	// store density
	density[gid] = rho;
	// density[gid] = flag;
#endif

}
